#include "hip/hip_runtime.h"
#include <fstream>
#include <iostream>
#include "../include/image_processing.hpp"
// #include "../include/labeling_algorithms.h"
// #include "../include/labels_solver.h"
#include "../include/two_pass_algorithm.hpp"

namespace gpu {

dim3 const block_size(16, 8);
dim3 const grid_size((WIDTH + block_size.x - 1) / block_size.x,
                     (HEIGHT + block_size.y - 1) / block_size.y);

const size_t frame_size = WIDTH * HEIGHT * sizeof(uint8_t);
uint8_t* device_frame;
uint8_t* device_temp_frame;

const int struct_elem_size = 2;
const int diameter = 2 * struct_elem_size + 1;
__constant__ uint8_t d_structuring_element[diameter * diameter];
__constant__ int d_struct_elem_size;

std::pair<int32_t, int32_t> laser_position;

void create_structuring_element(uint8_t* struct_elem, int struct_elem_size) {
  int diameter = 2 * struct_elem_size + 1;

  for (int i = 0; i < diameter; i++) {
    for (int j = 0; j < diameter; j++) {
      int y = i - struct_elem_size;  // y-coordinate relative to the center
      int x = j - struct_elem_size;  // x-coordinate relative to the center
      struct_elem[i * diameter + j] =
          (x * x + y * y <= struct_elem_size * struct_elem_size) ? 1 : 0;
    }
  }
}

void initialize_struct_elem() {
  uint8_t host_struct_elem[diameter * diameter];
  create_structuring_element(host_struct_elem, struct_elem_size);

  // Copy the host array to the GPU constant memory
  hipMemcpyToSymbol(HIP_SYMBOL(d_structuring_element), host_struct_elem,
                     diameter * diameter * sizeof(uint8_t));
  hipMemcpyToSymbol(HIP_SYMBOL(d_struct_elem_size), &struct_elem_size, sizeof(int));
}

void init_gpu() {
  hipMemcpyToSymbol(HIP_SYMBOL(d_WIDTH), &WIDTH, sizeof(int));
  hipMemcpyToSymbol(HIP_SYMBOL(d_HEIGHT), &HEIGHT, sizeof(int));

  hipMalloc((void**)&device_frame, frame_size);
  hipMalloc((void**)&device_temp_frame, frame_size);

  // init_structuring_element();
  initialize_struct_elem();
}

void free_gpu() {
  hipFree(device_frame);
  hipFree(device_temp_frame);
}

__global__ void gaussian_smoothing(uint8_t* input,
                                   uint8_t* output,
                                   int kernel_size,
                                   float sigma) {
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;

  if (x < d_WIDTH && y < d_HEIGHT) {
    float sum = 0.0f;
    float total_weight = 0.0f;
    int half_kernel_size = kernel_size / 2;

    for (int i = -half_kernel_size; i <= half_kernel_size; ++i) {
      for (int j = -half_kernel_size; j <= half_kernel_size; ++j) {
        int current_x = x + j;
        int current_y = y + i;

        if (current_x >= 0 && current_x < d_WIDTH && current_y >= 0 &&
            current_y < d_HEIGHT) {
          float weight = exp(-(i * i + j * j) / (2.0f * sigma * sigma));
          sum += input[current_y * d_WIDTH + current_x] * weight;
          total_weight += weight;
        }
      }
    }

    output[y * d_WIDTH + x] = static_cast<uint8_t>(sum / total_weight);
  }
}

__global__ void binarise(uint8_t* device_frame, uint8_t threshold) {
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;

  if (x < d_WIDTH && y < d_HEIGHT) {
    device_frame[y * d_WIDTH + x] =
        device_frame[y * d_WIDTH + x] >= threshold ? 255 : 0;
  }
}

__global__ void erosion(uint8_t* input, uint8_t* output) {
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;

  if (x < d_WIDTH && y < d_HEIGHT) {
    int min_val = 255;
    for (int i = -d_struct_elem_size; i <= d_struct_elem_size; i++) {
      for (int j = -d_struct_elem_size; j <= d_struct_elem_size; j++) {
        if (y + i >= 0 && y + i < d_HEIGHT && x + j >= 0 && x + j < d_WIDTH) {
          if (d_structuring_element[(i + d_struct_elem_size) *
                                        (2 * d_struct_elem_size + 1) +
                                    j + d_struct_elem_size] == 1) {
            int idx = (y + i) * d_WIDTH + (x + j);
            min_val = min(min_val, (int)input[idx]);
          }
        }
      }
    }
    output[y * d_WIDTH + x] = min_val;
  }
}

__global__ void dilation(uint8_t* input, uint8_t* output) {
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;

  if (x < d_WIDTH && y < d_HEIGHT) {
    int maxVal = 0;
    for (int i = -d_struct_elem_size; i <= d_struct_elem_size; i++) {
      for (int j = -d_struct_elem_size; j <= d_struct_elem_size; j++) {
        if (y + i >= 0 && y + i < d_HEIGHT && x + j >= 0 && x + j < d_WIDTH) {
          if (d_structuring_element[(i + d_struct_elem_size) *
                                        (2 * d_struct_elem_size + 1) +
                                    j + d_struct_elem_size] == 1) {
            int idx = (y + i) * d_WIDTH + (x + j);
            maxVal = max(maxVal, (int)input[idx]);
          }
        }
      }
    }
    output[y * d_WIDTH + x] = maxVal;
  }
}

bool is_blob_in_ignore_region(
    const std::pair<uint16_t, uint16_t>& blob,
    const std::pair<uint16_t, uint16_t>& ignore_region_top_left,
    const std::pair<uint16_t, uint16_t>& ignore_region_bottom_right) {
  uint16_t x = blob.first;
  uint16_t y = blob.second;
  return x >= ignore_region_top_left.first &&
         x <= ignore_region_bottom_right.first &&
         y >= ignore_region_top_left.second &&
         y <= ignore_region_bottom_right.second;
}

std::pair<int32_t, int32_t> distinguish_laser(
    std::vector<std::pair<uint16_t, uint16_t>> blobs,
    std::pair<uint16_t, uint16_t> camera_origin,
    std::pair<uint16_t, uint16_t> ignore_region_top_left,
    std::pair<uint16_t, uint16_t> ignore_region_bottom_right) {
  // std::cout << "All Blobs:" << std::endl;
  // for (size_t i = 0; i < blobs.size(); ++i) {
  //   std::cout << "Blob " << i + 1 << ": (" << blobs[i].first << ", "
  //             << blobs[i].second << ")" << std::endl;
  // }

  if (blobs.size() == 1)
    return blobs.at(0);

  std::pair<int32_t, int32_t> result = std::make_pair(-1, -1);
  double minDist = std::numeric_limits<double>::infinity();
  double maxDist = -1;

  uint16_t ox = camera_origin.first;
  uint16_t oy = camera_origin.second;

  for (size_t i = 0; i < blobs.size(); i++) {
    std::pair<uint16_t, uint16_t> blob = blobs[i];
    uint16_t x = blob.first;
    uint16_t y = blob.second;

    if (is_blob_in_ignore_region(blob, ignore_region_top_left,
                                 ignore_region_bottom_right)) {
      std::cout << "blob in ignore region" << std::endl;
      continue;  // Skip blobs in the ignore region
    }

    double dist = std::hypot(x - ox, y - oy);

    if (y <= oy && dist < minDist) {
      minDist = dist;
      result = std::make_pair(x, y);
    }

    if (y >= oy && dist > maxDist) {
      maxDist = dist;
      result = std::make_pair(x, y);
    }

    if (y > oy && result.first == -1)
      result = std::make_pair(x, y);
  }

  // std::cout << "result = (" << result.first << ", " << result.second << ")"
  //           << std::endl;
  return result;
}

std::pair<int32_t, int32_t> distinguish_laser(
    std::vector<std::pair<uint16_t, uint16_t>> blobs,
    std::pair<uint16_t, uint16_t> camera_origin) {
  if (blobs.size() < 1)
    return blobs.at(0);

  std::pair<uint16_t, uint16_t> blob1 = blobs[0];
  std::pair<uint16_t, uint16_t> blob2 = blobs[1];

  uint16_t ox = camera_origin.first;
  uint16_t oy = camera_origin.second;

  uint16_t x1 = blob1.first;
  uint16_t y1 = blob1.second;
  uint16_t x2 = blob2.first;
  uint16_t y2 = blob2.second;

  // When both blobs are at or below the camera origin the closest blob is the
  // laser
  if (y1 <= oy && y2 <= oy) {
    double dist1 = std::hypot(x1 - ox, y1 - oy);
    double dist2 = std::hypot(x2 - ox, y2 - oy);

    if (dist1 < dist2)
      return std::make_pair(x1, y1);
    else
      return std::make_pair(x2, y2);
  }

  // When both blobs are at or above the camera origin the furthest blob is the
  // laser
  if (y1 >= oy && y2 >= oy) {
    double dist1 = std::hypot(x1 - ox, y1 - oy);
    double dist2 = std::hypot(x2 - ox, y2 - oy);

    if (dist1 > dist2)
      return std::make_pair(x1, y1);
    else
      return std::make_pair(x2, y2);
  }

  // When blobs are on either side of the camera origin the laser is the blob
  // that is above the camera origin
  if (y1 > oy)
    return std::make_pair(x1, y1);
  else
    return std::make_pair(x2, y2);
}

void save_frame_to_text_file(const uint8_t* frame,
                             size_t size,
                             const std::string& filename) {
  std::ofstream out_file(filename);

  out_file << "{";
  if (!out_file) {
    std::cerr << "Could not open file for writing: " << filename << std::endl;
    return;
  }

  for (size_t i = 0; i < size; ++i) {
    out_file << static_cast<unsigned int>(
        frame[i]);  // Cast to unsigned int for proper text output
    if (i < size - 1) {
      out_file << ", ";  // Separate each byte by space
    }
  }
  out_file << "}";

  out_file.close();
  std::cout << "Saved frame to: " << filename << std::endl;
}

std::pair<int32_t, int32_t> detect_laser(uint8_t* red_frame,
                                         uint8_t threshold) {
  hipError_t err;

  err = hipMemcpy(device_frame, red_frame, frame_size, hipMemcpyHostToDevice);
  if (err != hipSuccess) {
    printf("CUDA error 1: %s\n", hipGetErrorString(err));
  }

  gaussian_smoothing<<<grid_size, block_size>>>(device_frame, device_temp_frame,
                                                5, 6.0f);
  err = hipMemcpy(device_frame, device_temp_frame, frame_size,
                   hipMemcpyDeviceToDevice);
  if (err != hipSuccess) {
    printf("CUDA error 2: %s\n", hipGetErrorString(err));
  }
  // hipDeviceSynchronize();

  binarise<<<grid_size, block_size>>>(device_frame, threshold);
  close_and_open();

  err = hipMemcpy(red_frame, device_frame, frame_size, hipMemcpyDeviceToHost);
  if (err != hipSuccess) {
    printf("CUDA error 3: %s\n", hipGetErrorString(err));
  }
  // save_frame_to_text_file(red_frame, WIDTH * HEIGHT, "gpu_frame.txt");

  std::chrono::high_resolution_clock::time_point start_time =
      std::chrono::high_resolution_clock::now();

  laser_position = distinguish_laser(
      find_blobs(red_frame), std::make_pair(X_ORIGIN_PX, Y_ORIGIN_PX),
      std::make_pair(0, 0), std::make_pair(0, 0));

  std::chrono::high_resolution_clock::time_point end_time =
      std::chrono::high_resolution_clock::now();
  uint32_t duration = std::chrono::duration_cast<std::chrono::microseconds>(
                          end_time - start_time)
                          .count();
  // std::cout << "GPU processing time = " << duration << " us" << std::endl;

  cv::Mat mat(HEIGHT, WIDTH, CV_8UC1, red_frame);
  cv::putText(mat,
              "laser pos = (" + std::to_string(laser_position.first) + ", " +
                  std::to_string(laser_position.second) + ")",
              cv::Point(10, 30), cv::FONT_HERSHEY_SIMPLEX, 1,
              cv::Scalar(255, 255, 255), 2);
  cv::imshow("pre-processed frame", mat);
  cv::waitKey(1);
  // std::cout << "laser pos = (" << laser_position.first << ", "
  //           << laser_position.second << ")" << std::endl;
  return laser_position;
  // return std::make_pair(-2, -2);
}

void opening() {
  erosion<<<grid_size, block_size>>>(device_frame, device_temp_frame);
  dilation<<<grid_size, block_size>>>(device_temp_frame, device_frame);
}

void closing() {
  dilation<<<grid_size, block_size>>>(device_frame, device_temp_frame);
  erosion<<<grid_size, block_size>>>(device_temp_frame, device_frame);
}

void open_and_close() {
  opening();
  closing();
}

void close_and_open() {
  closing();
  opening();
}

}  // namespace gpu
