#include "hip/hip_runtime.h"
#include <iostream>
#include "../include/image_processing.hpp"

namespace gpu {

typedef struct pt_ {
  int x, y;
} pt;

typedef struct blob_ {
  int min_x, max_x;
  int min_y, max_y;
  int cen_x, cen_y;
  int n_pixels;
  int ID;
} blob;

dim3 const block_size(16, 8);
dim3 const grid_size((WIDTH + block_size.x - 1) / block_size.x,
                     (HEIGHT + block_size.y - 1) / block_size.y);

const size_t frame_size = WIDTH * HEIGHT * sizeof(uint8_t);
uint8_t* device_frame;
uint8_t* device_temp_frame;

const int struct_elem_size = 2;
const int diameter = 2 * struct_elem_size + 1;
__constant__ uint8_t d_structuring_element[diameter * diameter];
__constant__ int d_struct_elem_size;

std::pair<int32_t, int32_t> laser_position;

void create_structuring_element(uint8_t* struct_elem, int struct_elem_size) {
  int diameter = 2 * struct_elem_size + 1;

  for (int i = 0; i < diameter; i++) {
    for (int j = 0; j < diameter; j++) {
      int y = i - struct_elem_size;  // y-coordinate relative to the center
      int x = j - struct_elem_size;  // x-coordinate relative to the center
      struct_elem[i * diameter + j] =
          (x * x + y * y <= struct_elem_size * struct_elem_size) ? 1 : 0;
    }
  }
}

void initialize_struct_elem() {
  uint8_t host_struct_elem[diameter * diameter];
  create_structuring_element(host_struct_elem, struct_elem_size);

  // Copy the host array to the GPU constant memory
  hipMemcpyToSymbol(HIP_SYMBOL(d_structuring_element), host_struct_elem,
                     diameter * diameter * sizeof(uint8_t));
  hipMemcpyToSymbol(HIP_SYMBOL(d_struct_elem_size), &struct_elem_size, sizeof(int));
}

void init_gpu() {
  hipMemcpyToSymbol(HIP_SYMBOL(d_WIDTH), &WIDTH, sizeof(int));
  hipMemcpyToSymbol(HIP_SYMBOL(d_HEIGHT), &HEIGHT, sizeof(int));

  hipMalloc((void**)&device_frame, frame_size);
  hipMalloc((void**)&device_temp_frame, frame_size);

  initialize_struct_elem();
}

void free_gpu() {
  hipFree(device_frame);
  hipFree(device_temp_frame);
}

__global__ void gaussian_smoothing(uint8_t* input,
                                   uint8_t* output,
                                   int kernel_size,
                                   float sigma) {
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;

  if (x < d_WIDTH && y < d_HEIGHT) {
    float sum = 0.0f;
    float total_weight = 0.0f;
    int half_kernel_size = kernel_size / 2;

    for (int i = -half_kernel_size; i <= half_kernel_size; ++i) {
      for (int j = -half_kernel_size; j <= half_kernel_size; ++j) {
        int current_x = x + j;
        int current_y = y + i;

        if (current_x >= 0 && current_x < d_WIDTH && current_y >= 0 &&
            current_y < d_HEIGHT) {
          float weight = exp(-(i * i + j * j) / (2.0f * sigma * sigma));
          sum += input[current_y * d_WIDTH + current_x] * weight;
          total_weight += weight;
        }
      }
    }

    output[y * d_WIDTH + x] = static_cast<uint8_t>(sum / total_weight);
  }
}

__global__ void binarise(uint8_t* device_frame, uint8_t threshold) {
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;

  if (x < d_WIDTH && y < d_HEIGHT) {
    device_frame[y * d_WIDTH + x] =
        device_frame[y * d_WIDTH + x] >= threshold ? 255 : 0;
  }
}

__global__ void erosion(uint8_t* input, uint8_t* output) {
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;

  if (x < d_WIDTH && y < d_HEIGHT) {
    int min_val = 255;
    for (int i = -d_struct_elem_size; i <= d_struct_elem_size; i++) {
      for (int j = -d_struct_elem_size; j <= d_struct_elem_size; j++) {
        if (y + i >= 0 && y + i < d_HEIGHT && x + j >= 0 && x + j < d_WIDTH) {
          if (d_structuring_element[(i + d_struct_elem_size) *
                                        (2 * d_struct_elem_size + 1) +
                                    j + d_struct_elem_size] == 1) {
            int idx = (y + i) * d_WIDTH + (x + j);
            min_val = min(min_val, (int)input[idx]);
          }
        }
      }
    }
    output[y * d_WIDTH + x] = min_val;
  }
}

__global__ void dilation(uint8_t* input, uint8_t* output) {
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;

  if (x < d_WIDTH && y < d_HEIGHT) {
    int maxVal = 0;
    for (int i = -d_struct_elem_size; i <= d_struct_elem_size; i++) {
      for (int j = -d_struct_elem_size; j <= d_struct_elem_size; j++) {
        if (y + i >= 0 && y + i < d_HEIGHT && x + j >= 0 && x + j < d_WIDTH) {
          if (d_structuring_element[(i + d_struct_elem_size) *
                                        (2 * d_struct_elem_size + 1) +
                                    j + d_struct_elem_size] == 1) {
            int idx = (y + i) * d_WIDTH + (x + j);
            maxVal = max(maxVal, (int)input[idx]);
          }
        }
      }
    }
    output[y * d_WIDTH + x] = maxVal;
  }
}

bool is_blob_in_ignore_region(
    const std::pair<uint16_t, uint16_t>& blob,
    const std::pair<uint16_t, uint16_t>& ignore_region_top_left,
    const std::pair<uint16_t, uint16_t>& ignore_region_bottom_right) {
  uint16_t x = blob.first;
  uint16_t y = blob.second;
  return x >= ignore_region_top_left.first &&
         x <= ignore_region_bottom_right.first &&
         y >= ignore_region_top_left.second &&
         y <= ignore_region_bottom_right.second;
}

std::pair<uint16_t, uint16_t> distinguish_laser(
    const std::vector<blob>& blobs,
    const std::pair<uint16_t, uint16_t> camera_origin,
    const std::pair<uint16_t, uint16_t> ignore_region_top_left,
    const std::pair<uint16_t, uint16_t> ignore_region_bottom_right) {
  if (blobs.size() == 1)
    return std::make_pair(blobs.at(0).cen_x, blobs.at(0).cen_y);

  std::pair<int32_t, int32_t> result = std::make_pair(-1, -1);
  double minDist = std::numeric_limits<double>::infinity();
  double maxDist = -1;

  uint16_t ox = camera_origin.first;
  uint16_t oy = camera_origin.second;

  for (size_t i = 0; i < blobs.size(); i++) {
    uint16_t x = blobs[i].cen_x;
    uint16_t y = blobs[i].cen_y;

    if (is_blob_in_ignore_region(std::make_pair(x, y), ignore_region_top_left,
                                 ignore_region_bottom_right)) {
      std::cout << "blob in ignore region" << std::endl;
      continue;  // Skip blobs in the ignore region
    }

    double dist = std::hypot(x - ox, y - oy);

    if (y <= oy && dist < minDist) {
      minDist = dist;
      result = std::make_pair(x, y);
    }

    if (y >= oy && dist > maxDist) {
      maxDist = dist;
      result = std::make_pair(x, y);
    }

    if (y > oy && result.first == -1)
      result = std::make_pair(x, y);
  }

  return result;
}

void get_blobs(uint8_t* frame, std::vector<blob>& blobs) {
  // int i, j, k, l, r = img.rows, c = img.cols, id = 1;
  int i, j, k, l, r = HEIGHT, c = WIDTH, id = 1;
  std::vector<std::vector<int>> pixel_ID(r, std::vector<int>(c, -1));
  // Stores ID of a pixel; -1 means unvisited
  std::queue<pt> open_list;
  // Breadth-First-Search hence queue of points
  for (i = 1; i < r - 1; i++) {
    for (j = 1; j < c - 1; j++) {
      if (frame[i * WIDTH + j] == 0 || pixel_ID[i][j] > -1) {
        continue;
      }
      pt start = {j, i};
      open_list.push(start);
      int sum_x = 0, sum_y = 0, n_pixels = 0, max_x = 0, max_y = 0;
      int min_x = c + 1, min_y = r + 1;
      while (!open_list.empty()) {  // Dequeue the element at the head of the
                                    // queue
        pt top = open_list.front();
        open_list.pop();
        pixel_ID[top.y][top.x] = id;
        n_pixels++;  // To obtain the bounding box of the blob w.r.t the
                     // original image
        min_x = (top.x < min_x) ? top.x : min_x;
        min_y = (top.y < min_y) ? top.y : min_y;
        max_x = (top.x > max_x) ? top.x : max_x;
        max_y = (top.y > max_y) ? top.y : max_y;
        sum_y += top.y;
        sum_x += top.x;  // Add the 8-connected neighbours that are yet to be
                         // visited, to the queue
        for (k = top.y - 1; k <= top.y + 1; k++) {
          for (l = top.x - 1; l <= top.x + 1; l++) {
            if (frame[k * WIDTH + l] == 0 || pixel_ID[k][l] > -1) {
              continue;
            }
            pt next = {l, k};
            pixel_ID[k][l] = id;
            open_list.push(next);
          }
        }
      }

      if (n_pixels < 20) {  // At least 20 pixels
        continue;
      }

      blob nextcentre = {
          min_x,    max_x, min_y, max_y, sum_x / n_pixels, sum_y / n_pixels,
          n_pixels, id};
      blobs.push_back(nextcentre);
      id++;
    }
  }
  std::cout
      << blobs.size();  // To test correctness; can use the vector as desired
}

void get_blobs(cv::Mat frame, std::vector<blob>& blobs) {
  // int i, j, k, l, r = img.rows, c = img.cols, id = 1;
  int i, j, k, l, r = HEIGHT, c = WIDTH, id = 1;
  std::vector<std::vector<int>> pixel_ID(r, std::vector<int>(c, -1));
  // Stores ID of a pixel; -1 means unvisited
  std::queue<pt> open_list;
  // Breadth-First-Search hence queue of points
  for (i = 1; i < r - 1; i++) {
    for (j = 1; j < c - 1; j++) {
      if (frame.at<uint8_t>(i, j) == 0 || pixel_ID[i][j] > -1) {
        // if (frame[i * WIDTH + j] == 0 || pixel_ID[i][j] > -1) {
        continue;
      }
      pt start = {j, i};
      open_list.push(start);
      int sum_x = 0, sum_y = 0, n_pixels = 0, max_x = 0, max_y = 0;
      int min_x = c + 1, min_y = r + 1;
      while (!open_list.empty()) {  // Dequeue the element at the head of the
                                    // queue
        pt top = open_list.front();
        open_list.pop();
        pixel_ID[top.y][top.x] = id;
        n_pixels++;  // To obtain the bounding box of the blob w.r.t the
                     // original image
        min_x = (top.x < min_x) ? top.x : min_x;
        min_y = (top.y < min_y) ? top.y : min_y;
        max_x = (top.x > max_x) ? top.x : max_x;
        max_y = (top.y > max_y) ? top.y : max_y;
        sum_y += top.y;
        sum_x += top.x;  // Add the 8-connected neighbours that are yet to be
                         // visited, to the queue
        for (k = top.y - 1; k <= top.y + 1; k++) {
          for (l = top.x - 1; l <= top.x + 1; l++) {
            if (frame.at<uint8_t>(k, l) == 0 || pixel_ID[k][l] > -1) {
              // if (frame[k * WIDTH + l] == 0 || pixel_ID[k][l] > -1) {
              continue;
            }
            pt next = {l, k};
            pixel_ID[k][l] = id;
            open_list.push(next);
          }
        }
      }

      if (n_pixels < 20) {  // At least 20 pixels
        continue;
      }

      blob nextcentre = {
          min_x,    max_x, min_y, max_y, sum_x / n_pixels, sum_y / n_pixels,
          n_pixels, id};
      blobs.push_back(nextcentre);
      id++;
    }
  }
  std::cout
      << blobs.size();  // To test correctness; can use the vector as desired
}

std::pair<int32_t, int32_t> detect_laser(uint8_t* red_frame,
                                         uint8_t threshold) {
  hipError_t err;

  err = hipMemcpy(device_frame, red_frame, frame_size, hipMemcpyHostToDevice);
  if (err != hipSuccess) {
    printf("CUDA error 1: %s\n", hipGetErrorString(err));
  }

  gaussian_smoothing<<<grid_size, block_size>>>(device_frame, device_temp_frame,
                                                5, 6.0f);
  err = hipMemcpy(device_frame, device_temp_frame, frame_size,
                   hipMemcpyDeviceToDevice);
  if (err != hipSuccess) {
    printf("CUDA error 2: %s\n", hipGetErrorString(err));
  }
  // hipDeviceSynchronize();

  binarise<<<grid_size, block_size>>>(device_frame, threshold);
  close_and_open();

  err = hipMemcpy(red_frame, device_frame, frame_size, hipMemcpyDeviceToHost);
  if (err != hipSuccess) {
    printf("CUDA error 3: %s\n", hipGetErrorString(err));
  }

  std::vector<blob> blobs;
  get_blobs(red_frame, blobs);
  laser_position =
      distinguish_laser(blobs, std::make_pair(X_ORIGIN_PX, Y_ORIGIN_PX),
                        std::make_pair(0, 0), std::make_pair(0, 0));

  cv::Mat mat(HEIGHT, WIDTH, CV_8UC1, red_frame);
  cv::putText(mat,
              "laser pos = (" + std::to_string(laser_position.first) + ", " +
                  std::to_string(laser_position.second) + ")",
              cv::Point(10, 30), cv::FONT_HERSHEY_SIMPLEX, 1,
              cv::Scalar(255, 255, 255), 2);
  cv::imshow("pre-processed frame", mat);
  cv::waitKey(1);
  return laser_position;
}

std::pair<int32_t, int32_t> detect_laser(cv::Mat red_frame, uint8_t threshold) {
  // hipError_t err;

  // Upload the CPU input to the GPU
  cv::cuda::GpuMat input_gpu_mat;
  input_gpu_mat.upload(red_frame);
  // Create an output GpuMat
  cv::cuda::GpuMat output_gpu_mat(red_frame.size(), red_frame.type());

  gaussian_smoothing<<<grid_size, block_size>>>(
      input_gpu_mat.ptr<uint8_t>(), output_gpu_mat.ptr<uint8_t>(), 5, 6.0f);
  // err = hipMemcpy(device_frame, device_temp_frame, frame_size,
  //                  hipMemcpyDeviceToDevice);
  // if (err != hipSuccess) {
  //   printf("CUDA error 2: %s\n", hipGetErrorString(err));
  // }
  // hipDeviceSynchronize();

  binarise<<<grid_size, block_size>>>(output_gpu_mat.ptr<uint8_t>(), threshold);
  close_and_open();

  // err = hipMemcpy(red_frame, device_frame, frame_size,
  // hipMemcpyDeviceToHost); if (err != hipSuccess) {
  //   printf("CUDA error 3: %s\n", hipGetErrorString(err));
  // }

  // Download the result back to CPU
  output_gpu_mat.download(red_frame);
  std::vector<blob> blobs;
  get_blobs(red_frame, blobs);
  laser_position =
      distinguish_laser(blobs, std::make_pair(X_ORIGIN_PX, Y_ORIGIN_PX),
                        std::make_pair(0, 0), std::make_pair(0, 0));

  // cv::Mat mat(HEIGHT, WIDTH, CV_8UC1, red_frame);
  cv::putText(red_frame,
              "laser pos = (" + std::to_string(laser_position.first) + ", " +
                  std::to_string(laser_position.second) + ")",
              cv::Point(10, 30), cv::FONT_HERSHEY_SIMPLEX, 1,
              cv::Scalar(255, 255, 255), 2);
  cv::imshow("pre-processed frame", red_frame);
  cv::waitKey(1);
  return laser_position;
}

void opening() {
  erosion<<<grid_size, block_size>>>(device_frame, device_temp_frame);
  dilation<<<grid_size, block_size>>>(device_temp_frame, device_frame);
}

void opening(uint8_t* input, uint8_t* output) {
  erosion<<<grid_size, block_size>>>(input, output);
  dilation<<<grid_size, block_size>>>(output, input);
}

void closing() {
  dilation<<<grid_size, block_size>>>(device_frame, device_temp_frame);
  erosion<<<grid_size, block_size>>>(device_temp_frame, device_frame);
}

void closing(uint8_t* input, uint8_t* output) {
  dilation<<<grid_size, block_size>>>(output, input);
  erosion<<<grid_size, block_size>>>(input, output);
}

void open_and_close() {
  opening();
  closing();
}

void close_and_open() {
  closing();
  opening();
}

std::vector<blob> detect_mosquitoes(cv::Mat red_frame, uint8_t threshold) {
  cv::cuda::GpuMat input_gpu_mat;
  input_gpu_mat.upload(red_frame);
  cv::cuda::GpuMat output_gpu_mat(red_frame.size(), red_frame.type());

  gaussian_smoothing<<<grid_size, block_size>>>(
      input_gpu_mat.ptr<uint8_t>(), output_gpu_mat.ptr<uint8_t>(), 5, 6.0f);
  binarise<<<grid_size, block_size>>>(output_gpu_mat.ptr<uint8_t>(), threshold);
  closing(output_gpu_mat.ptr<uint8_t>(), input_gpu_mat.ptr<uint8_t>());
  opening(input_gpu_mat.ptr<uint8_t>(), output_gpu_mat.ptr<uint8_t>());

  output_gpu_mat.download(red_frame);
  std::vector<blob> blobs;
  get_blobs(red_frame, blobs);

  cv::imshow("mosquitoes pre-processed", red_frame);
  cv::waitKey(1);
  return blobs;
}

__global__ void subtract_background(uint8_t* device_frame) {
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;

  if (x < d_WIDTH && y < d_HEIGHT) {
    background[y * d_WIDTH + x] =
        learning_rate * device_frame[y * d_WIDTH + x] +
        (1 - learning_rate) * background[y * d_WIDTH + x];
    device_frame[y * d_WIDTH + x] =
        device_frame[y * d_WIDTH + x] - background[y * d_WIDTH + x];
  }
}

class Subtractor {
 private:
  __constant__ cv::cuda::GpuMat background;
  __constant__ float learning_rate;

 public:
  Subtractor(cv::Mat backgroud, float learning_rate)
      : learning_rate(learning_rate) {
    this->background.upload(backgroud);
  }

  std::vector<blob> detect_mosquitoes(cv::Mat red_frame) {
    cv::cuda::GpuMat input_gpu_mat;
    input_gpu_mat.upload(red_frame);
    cv::cuda::GpuMat output_gpu_mat(red_frame.size(), red_frame.type());

    gaussian_smoothing<<<grid_size, block_size>>>(
        input_gpu_mat.ptr<uint8_t>(), output_gpu_mat.ptr<uint8_t>(), 5, 6.0f);
    // binarise<<<grid_size, block_size>>>(output_gpu_mat.ptr<uint8_t>(),
    // threshold);
    subtract_background<<<grid_size, block_size>>>(
        output_gpu_mat.ptr<uint8_t>());
    closing(output_gpu_mat.ptr<uint8_t>(), input_gpu_mat.ptr<uint8_t>());
    opening(input_gpu_mat.ptr<uint8_t>(), output_gpu_mat.ptr<uint8_t>());

    output_gpu_mat.download(red_frame);
    std::vector<blob> blobs;
    get_blobs(red_frame, blobs);

    cv::imshow("mosquitoes pre-processed", red_frame);
    cv::waitKey(1);
    return blobs;
  }
}

}  // namespace gpu
