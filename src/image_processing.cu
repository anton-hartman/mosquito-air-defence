#include "hip/hip_runtime.h"
#include <iostream>
#include "../include/image_processing.hpp"

namespace gpu {

dim3 const block_size(16, 8);
dim3 const grid_size((WIDTH + block_size.x - 1) / block_size.x,
                     (HEIGHT + block_size.y - 1) / block_size.y);

const size_t frame_size = WIDTH * HEIGHT * sizeof(uint8_t);
uint8_t* device_frame;
uint8_t* device_temp_frame;

// For erosion and dilation
// uint8_t* struct_elem_size = new uint8_t(1);
uint8_t struct_elem_size = 1;
// uint8_t* d_struct_elem_size;
// uint8_t* d_structuring_element;

uint8_t* create_structuring_element() {
  int diameter = 2 * (struct_elem_size) + 1;
  uint8_t* struct_elem = new uint8_t[diameter * diameter];

  for (int i = 0; i < diameter; i++) {
    for (int j = 0; j < diameter; j++) {
      int y = i - (struct_elem_size);  // y-coordinate relative to the center
      int x = j - (struct_elem_size);  // x-coordinate relative to the center
      struct_elem[i * diameter + j] =
          (x * x + y * y <= (struct_elem_size) * (struct_elem_size)) ? 1 : 0;
    }
  }
  return struct_elem;
}

size_t sizeof_structuring_elem() {
  int diameter = 2 * (struct_elem_size) + 1;
  return (diameter * diameter) * sizeof(uint8_t);
}

void init_structuring_element() {
  std::cout << "Struct elem: " << *create_structuring_element() << std::endl;
  hipMalloc((void**)&d_struct_elem_size, sizeof(uint8_t));
  hipMemcpy(&d_struct_elem_size, &struct_elem_size, sizeof(uint8_t),
             hipMemcpyHostToDevice);

  hipMalloc((void**)&d_structuring_element, sizeof_structuring_elem());
  hipMemcpy(d_structuring_element, create_structuring_element(),
             sizeof_structuring_elem(), hipMemcpyHostToDevice);
}

void init_gpu() {
  hipMemcpyToSymbol(HIP_SYMBOL(d_WIDTH), &WIDTH, sizeof(int));
  hipMemcpyToSymbol(HIP_SYMBOL(d_HEIGHT), &HEIGHT, sizeof(int));

  hipMalloc((void**)&device_frame, frame_size);
  hipMalloc((void**)&device_temp_frame, frame_size);

  init_structuring_element();
}

void free_gpu() {
  hipFree(device_frame);
  hipFree(device_temp_frame);
  // delete[] d_structuring_element;
}

// void undistort(cv::Mat& input_frame,
//                cv::Mat& output_frame,
//                cv::Mat& camera_matrix,
//                cv::Mat& dist_coeffs) {
//   cv::undistort(input_frame, output_frame, camera_matrix, dist_coeffs);
// }

__global__ void binarise(uint8_t* device_frame, uint8_t threshold) {
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;

  if (x < d_WIDTH && y < d_HEIGHT) {
    device_frame[y * d_WIDTH + x] =
        device_frame[y * d_WIDTH + x] >= threshold ? 255 : 0;
  }
}

__global__ void erosion(uint8_t* input, uint8_t* output) {
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;

  if (x < d_WIDTH && y < d_HEIGHT) {
    int min_val = 255;
    for (int i = -d_struct_elem_size; i <= d_struct_elem_size; i++) {
      for (int j = -d_struct_elem_size; j <= d_struct_elem_size; j++) {
        if (y + i >= 0 && y + i < d_HEIGHT && x + j >= 0 && x + j < d_WIDTH) {
          if (d_structuring_element[(i + d_struct_elem_size) *
                                        (2 * d_struct_elem_size + 1) +
                                    j + d_struct_elem_size] == 1) {
            int idx = (y + i) * d_WIDTH + (x + j);
            min_val = min(min_val, (int)input[idx]);
          }
        }
      }
    }
    output[y * d_WIDTH + x] = min_val;
  }
}

__global__ void dilation(uint8_t* input, uint8_t* output) {
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;

  if (x < d_WIDTH && y < d_HEIGHT) {
    int maxVal = 0;
    for (int i = -d_struct_elem_size; i <= d_struct_elem_size; i++) {
      for (int j = -d_struct_elem_size; j <= d_struct_elem_size; j++) {
        if (y + i >= 0 && y + i < d_HEIGHT && x + j >= 0 && x + j < d_WIDTH) {
          if (d_structuring_element[(i + d_struct_elem_size) *
                                        (2 * d_struct_elem_size + 1) +
                                    j + d_struct_elem_size] == 1) {
            int idx = (y + i) * d_WIDTH + (x + j);
            maxVal = max(maxVal, (int)input[idx]);
          }
        }
      }
    }
    output[y * d_WIDTH + x] = maxVal;
  }
}

uint32_t detect_laser(uint8_t* red_frame, uint8_t threshold) {
  hipError_t err;

  hipMemcpy(device_frame, red_frame, frame_size, hipMemcpyHostToDevice);

  binarise<<<grid_size, block_size>>>(device_frame, threshold);
  hipDeviceSynchronize();
  err = hipMemcpy(red_frame, device_frame, frame_size, hipMemcpyDeviceToHost);
  if (err != hipSuccess) {
    printf("CUDA error binarise: %s\n", hipGetErrorString(err));
  }
  // cv::Mat bin_mat(HEIGHT, WIDTH, CV_8UC1, red_frame);
  // cv::imshow("binarise", bin_mat);
  // cv::waitKey(1);

  // erosion<<<grid_size, block_size>>>(device_frame, device_temp_frame);
  // hipDeviceSynchronize();

  // err = hipMemcpy(red_frame, device_temp_frame, frame_size,
  //                  hipMemcpyDeviceToHost);
  // if (err != hipSuccess) {
  //   printf("CUDA error erosion: %s\n", hipGetErrorString(err));
  // }
  // cv::Mat erode_mat(HEIGHT, WIDTH, CV_8UC1, red_frame);
  // cv::imshow("erosion", erode_mat);
  // cv::waitKey(1);

  return 0;
}

// void opening() {
//   erosion<<<grid_size, block_size>>>(device_frame, device_temp_frame,
//                                      struct_elem, struct_elem_size);
//   dilation<<<grid_size, block_size>>>(device_temp_frame, device_frame,
//                                       struct_elem, struct_elem_size);
// }

}  // namespace gpu
