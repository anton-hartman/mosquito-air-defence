#include "hip/hip_runtime.h"
#include <chrono>
#include <iostream>
#include <thread>
#include "../include/image_processing.hpp"

namespace gpu {

std::pair<uint16_t, uint16_t> ignore_region_top_left = {523, 293};
std::pair<uint16_t, uint16_t> ignore_region_bottom_right = {553, 316};

void set_ignore_region(std::pair<uint16_t, uint16_t> top_left,
                       std::pair<uint16_t, uint16_t> bottom_right) {
  ignore_region_top_left = top_left;
  ignore_region_bottom_right = bottom_right;
}

dim3 const block_size(16, 8);
dim3 const grid_size((COLS + block_size.x - 1) / block_size.x,
                     (ROWS + block_size.y - 1) / block_size.y);

const size_t frame_size = COLS * ROWS * sizeof(uint8_t);
uint8_t* d_frame_1;
uint8_t* d_frame_2;
uint8_t* mos_d_frame_1;
uint8_t* mos_d_frame_2;

const int struct_elem_size = 2;
const int diameter = 2 * struct_elem_size + 1;
__constant__ uint8_t d_structuring_element[diameter * diameter];
__constant__ int d_struct_elem_size;

std::pair<int32_t, int32_t> laser_position = std::make_pair(-2, -2);

void create_structuring_element(uint8_t* struct_elem, int struct_elem_size) {
  int diameter = 2 * struct_elem_size + 1;

  for (int i = 0; i < diameter; i++) {
    for (int j = 0; j < diameter; j++) {
      int y = i - struct_elem_size;  // y-coordinate relative to the center
      int x = j - struct_elem_size;  // x-coordinate relative to the center
      struct_elem[i * diameter + j] =
          (x * x + y * y <= struct_elem_size * struct_elem_size) ? 1 : 0;
    }
  }
}

void initialize_struct_elem() {
  uint8_t host_struct_elem[diameter * diameter];
  create_structuring_element(host_struct_elem, struct_elem_size);

  // Copy the host array to the GPU constant memory
  hipMemcpyToSymbol(HIP_SYMBOL(d_structuring_element), host_struct_elem,
                     diameter * diameter * sizeof(uint8_t));
  hipMemcpyToSymbol(HIP_SYMBOL(d_struct_elem_size), &struct_elem_size, sizeof(int));
}

void init_gpu() {
  hipMemcpyToSymbol(HIP_SYMBOL(d_COLS), &COLS, sizeof(int));
  hipMemcpyToSymbol(HIP_SYMBOL(d_ROWS), &ROWS, sizeof(int));

  hipMalloc((void**)&d_frame_1, frame_size);
  hipMalloc((void**)&d_frame_2, frame_size);
  hipMalloc((void**)&mos_d_frame_1, frame_size);
  hipMalloc((void**)&mos_d_frame_2, frame_size);

  initialize_struct_elem();

  hipMemcpyToSymbol(HIP_SYMBOL(d_learning_rate), &learning_rate, sizeof(float));
  hipMalloc((void**)&d_background, frame_size);
}

void free_gpu() {
  hipFree(d_frame_1);
  hipFree(d_frame_2);
  hipFree(mos_d_frame_1);
  hipFree(mos_d_frame_2);
}

__global__ void gaussian_smoothing(uint8_t* input,
                                   uint8_t* output,
                                   int kernel_size,
                                   float sigma) {
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;

  if (x < d_COLS && y < d_ROWS) {
    float sum = 0.0f;
    float total_weight = 0.0f;
    int half_kernel_size = kernel_size / 2;

    for (int i = -half_kernel_size; i <= half_kernel_size; ++i) {
      for (int j = -half_kernel_size; j <= half_kernel_size; ++j) {
        int current_x = x + j;
        int current_y = y + i;

        if (current_x >= 0 && current_x < d_COLS && current_y >= 0 &&
            current_y < d_ROWS) {
          float weight = exp(-(i * i + j * j) / (2.0f * sigma * sigma));
          sum += input[current_y * d_COLS + current_x] * weight;
          total_weight += weight;
        }
      }
    }

    output[y * d_COLS + x] = static_cast<uint8_t>(sum / total_weight);
  }
}

__global__ void binarise_gt(uint8_t* device_frame, uint8_t threshold) {
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;

  if (x < d_COLS && y < d_ROWS) {
    device_frame[y * d_COLS + x] =
        device_frame[y * d_COLS + x] >= threshold ? 255 : 0;
  }
}

__global__ void binarise_lt(uint8_t* device_frame, uint8_t threshold) {
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;

  if (x < d_COLS && y < d_ROWS) {
    device_frame[y * d_COLS + x] =
        device_frame[y * d_COLS + x] <= threshold ? 255 : 0;
  }
}

__global__ void subtract_and_update_background(uint8_t* device_frame) {
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;

  if (x < d_COLS && y < d_ROWS) {
    d_background[y * d_COLS + x] =
        d_learning_rate * device_frame[y * d_COLS + x] +
        (1 - d_learning_rate) * d_background[y * d_COLS + x];
    device_frame[y * d_COLS + x] =
        abs(device_frame[y * d_COLS + x] - d_background[y * d_COLS + x]);
  }
}

__global__ void erosion(uint8_t* input, uint8_t* output) {
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;

  if (x < d_COLS && y < d_ROWS) {
    int min_val = 255;
    for (int i = -d_struct_elem_size; i <= d_struct_elem_size; i++) {
      for (int j = -d_struct_elem_size; j <= d_struct_elem_size; j++) {
        if (y + i >= 0 && y + i < d_ROWS && x + j >= 0 && x + j < d_COLS) {
          if (d_structuring_element[(i + d_struct_elem_size) *
                                        (2 * d_struct_elem_size + 1) +
                                    j + d_struct_elem_size] == 1) {
            int idx = (y + i) * d_COLS + (x + j);
            min_val = min(min_val, (int)input[idx]);
          }
        }
      }
    }
    output[y * d_COLS + x] = min_val;
  }
}

__global__ void dilation(uint8_t* input, uint8_t* output) {
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;

  if (x < d_COLS && y < d_ROWS) {
    int maxVal = 0;
    for (int i = -d_struct_elem_size; i <= d_struct_elem_size; i++) {
      for (int j = -d_struct_elem_size; j <= d_struct_elem_size; j++) {
        if (y + i >= 0 && y + i < d_ROWS && x + j >= 0 && x + j < d_COLS) {
          if (d_structuring_element[(i + d_struct_elem_size) *
                                        (2 * d_struct_elem_size + 1) +
                                    j + d_struct_elem_size] == 1) {
            int idx = (y + i) * d_COLS + (x + j);
            maxVal = max(maxVal, (int)input[idx]);
          }
        }
      }
    }
    output[y * d_COLS + x] = maxVal;
  }
}

bool is_blob_in_ignore_region(
    const std::pair<uint16_t, uint16_t>& blob,
    const std::pair<uint16_t, uint16_t>& ignore_region_top_left,
    const std::pair<uint16_t, uint16_t>& ignore_region_bottom_right) {
  uint16_t x = blob.first;
  uint16_t y = blob.second;
  return x >= ignore_region_top_left.first &&
         x <= ignore_region_bottom_right.first &&
         y >= ignore_region_top_left.second &&
         y <= ignore_region_bottom_right.second;
}

std::pair<uint16_t, uint16_t> distinguish_laser(
    const std::vector<Blob>& blobs) {
  if (blobs.size() == 1)
    return std::make_pair(blobs.at(0).cen_x, blobs.at(0).cen_y);

  std::pair<int32_t, int32_t> result = std::make_pair(-1, -1);
  double minDist = std::numeric_limits<double>::infinity();
  double maxDist = -1;

  for (size_t i = 0; i < blobs.size(); i++) {
    uint16_t x = blobs[i].cen_x;
    uint16_t y = blobs[i].cen_y;

    if (is_blob_in_ignore_region(std::make_pair(x, y), ignore_region_top_left,
                                 ignore_region_bottom_right)) {
      // std::cout << "blob in ignore region" << std::endl;
      continue;  // Skip blobs in the ignore region
    }

    double dist = std::hypot(x - C_X, y - C_Y);

    if (y <= C_Y && dist < minDist) {
      minDist = dist;
      result = std::make_pair(x, y);
    }

    if (y >= C_Y && dist > maxDist) {
      maxDist = dist;
      result = std::make_pair(x, y);
    }

    if (y > C_Y && result.first == -1)
      result = std::make_pair(x, y);
  }

  return result;
}

std::pair<uint16_t, uint16_t> distinguish_laser_only_2(
    const std::vector<Blob>& blobs) {
  if (blobs.size() == 1) {
    return std::make_pair(blobs.at(0).cen_x, blobs.at(0).cen_y);
  } else if (blobs.size() < 1) {
    return std::make_pair(-1, -1);
  }

  std::vector<Blob> two_blobs;
  for (size_t i = 0; i < blobs.size(); i++) {
    if (is_blob_in_ignore_region({blobs[i].cen_x, blobs[i].cen_y},
                                 ignore_region_top_left,
                                 ignore_region_bottom_right)) {
      continue;
    } else {
      two_blobs.push_back(blobs[i]);
    }
  }

  if (two_blobs.size() == 1) {
    return std::make_pair(two_blobs[0].cen_x, two_blobs[0].cen_y);
  } else if (two_blobs.size() < 1) {
    return std::make_pair(-2, -2);
  }

  if (two_blobs.size() > 2) {
    std::cout << "More than two blobs outside of ignore region: "
              << std::to_string(two_blobs.size()) << std::endl;
  }

  uint16_t x1 = two_blobs[0].cen_x;
  uint16_t y1 = two_blobs[0].cen_y;
  uint16_t x2 = two_blobs[1].cen_x;
  uint16_t y2 = two_blobs[1].cen_y;

  // When both blobs are at or below the camera origin, then the one closer to
  // the origin of the camera is the laser.
  if (y1 >= C_Y && y2 >= C_Y) {
    if (y1 < y2) {
      // y1 is closer to the camera origin
      return std::make_pair(x1, y1);
    } else {
      return std::make_pair(x2, y2);
    }
  }

  // When both blobs are at or above the camera origin, then the one farther
  // from the origin of the camera is the laser.
  if (y1 <= C_Y && y2 <= C_Y) {
    if (y1 < y2) {
      // y1 is further from the camera origin
      return std::make_pair(x1, y1);
    } else {
      return std::make_pair(x2, y2);
    }
  }

  // When blobs are on either side of the camera origin
  if (y1 < C_Y)
    return std::make_pair(x1, y1);
  else
    return std::make_pair(x2, y2);
}

int get_blobs(cv::Mat frame, std::vector<Blob>& blobs) {
  int i, j, k, l, r = frame.rows, c = frame.cols, id = 1;
  // Stores ID of a pixel; -1 means unvisited
  std::vector<std::vector<int>> pixel_ID(r, std::vector<int>(c, -1));
  std::queue<Pt> open_list;  // Breadth-First-Search hence queue of points

  for (i = 1; i < r - 1; i++) {
    for (j = 1; j < c - 1; j++) {
      if (i >= r || j >= c || frame.at<uint8_t>(i, j) == 0 ||
          pixel_ID[i][j] > -1) {
        continue;
      }
      Pt start = {j, i};
      open_list.push(start);
      int sum_x = 0, sum_y = 0, n_pixels = 0, max_x = 0, max_y = 0;
      int min_x = c + 1, min_y = r + 1;
      // Dequeue the element at the head of the queue
      while (!open_list.empty()) {
        Pt top = open_list.front();
        open_list.pop();
        if (top.y >= r || top.x >= c) {
          continue;
        }

        pixel_ID[top.y][top.x] = id;
        n_pixels++;
        // To obtain the bounding box of the blob w.r.t the original image
        min_x = (top.x < min_x) ? top.x : min_x;
        min_y = (top.y < min_y) ? top.y : min_y;
        max_x = (top.x > max_x) ? top.x : max_x;
        max_y = (top.y > max_y) ? top.y : max_y;
        sum_y += top.y;
        sum_x += top.x;

        // Add the 8 - connected neighbours that are yet to be visited, to the
        // queue
        for (k = top.y - 1; k <= top.y + 1; k++) {
          for (l = top.x - 1; l <= top.x + 1; l++) {
            if (k < 0 || l < 0 || k >= r || l >= c ||
                frame.at<uint8_t>(k, l) == 0 || pixel_ID[k][l] > -1) {
              continue;
            }
            Pt next = {l, k};
            pixel_ID[k][l] = id;
            open_list.push(next);
          }
        }
      }

      if (n_pixels < 20) {  // At least 20 pixels
        continue;
      }

      Blob nextcentre = {
          min_x,    max_x, min_y, max_y, sum_x / n_pixels, sum_y / n_pixels,
          n_pixels, id};
      blobs.push_back(nextcentre);
      id++;
    }
  }
  return blobs.size();
}

std::pair<int32_t, int32_t> detect_laser(cv::Mat red_frame, uint8_t threshold) {
  hipError_t err = hipMemcpy(d_frame_1, red_frame.ptr(), frame_size,
                               hipMemcpyHostToDevice);
  (err != hipSuccess) ? printf("CUDA err: %s\n", hipGetErrorString(err)) : 0;

  gaussian_smoothing<<<grid_size, block_size>>>(d_frame_1, d_frame_2, 5, 6.0f);
  binarise_gt<<<grid_size, block_size>>>(d_frame_2, threshold);
  dilation<<<grid_size, block_size>>>(d_frame_2, d_frame_1);
  erosion<<<grid_size, block_size>>>(d_frame_1, d_frame_2);
  erosion<<<grid_size, block_size>>>(d_frame_2, d_frame_1);
  dilation<<<grid_size, block_size>>>(d_frame_1, d_frame_2);

  err = hipMemcpy(red_frame.ptr(), d_frame_2, frame_size,
                   hipMemcpyDeviceToHost);
  (err != hipSuccess) ? printf("CUDA err: %s\n", hipGetErrorString(err)) : 0;

  int num_blobs = -2;
  std::vector<Blob> blobs;
  num_blobs = get_blobs(red_frame, blobs);
  laser_position = distinguish_laser_only_2(blobs);

  for (size_t i = 0; i < blobs.size(); i++) {
    cv::circle(red_frame, cv::Point(blobs[i].cen_x, blobs[i].cen_y), 20,
               cv::Scalar(150, 255, 255), 2);
    cv::putText(red_frame, std::to_string(i),
                cv::Point(blobs[i].cen_x + 10, blobs[i].cen_y + 10),
                cv::FONT_HERSHEY_SIMPLEX, 1, cv::Scalar(255, 255, 255), 1);
  }
  cv::putText(red_frame,
              "laser pos = (" + std::to_string(laser_position.first) + ", " +
                  std::to_string(laser_position.second) +
                  ")  num blobs = " + std::to_string(num_blobs),
              cv::Point(10, 30), cv::FONT_HERSHEY_SIMPLEX, 1,
              cv::Scalar(255, 255, 255), 2);
  cv::imshow("cv::Mat processed", red_frame);
  cv::waitKey(1);
  return laser_position;
}

std::vector<Pt> detect_mosquitoes(cv::Mat red_frame,
                                  uint8_t threshold,
                                  bool bg_sub) {
  hipError_t err = hipMemcpy(mos_d_frame_1, red_frame.ptr(), frame_size,
                               hipMemcpyHostToDevice);
  (err != hipSuccess) ? printf("CUDA err: %s\n", hipGetErrorString(err)) : 0;

  gaussian_smoothing<<<grid_size, block_size>>>(mos_d_frame_1, mos_d_frame_2, 5,
                                                6.0f);
  if (bg_sub) {
    subtract_and_update_background<<<grid_size, block_size>>>(mos_d_frame_2);
    binarise_gt<<<grid_size, block_size>>>(mos_d_frame_2, threshold);
  } else {
    binarise_lt<<<grid_size, block_size>>>(mos_d_frame_2, threshold);
  }
  dilation<<<grid_size, block_size>>>(mos_d_frame_2, mos_d_frame_1);
  erosion<<<grid_size, block_size>>>(mos_d_frame_1, mos_d_frame_2);
  erosion<<<grid_size, block_size>>>(mos_d_frame_2, mos_d_frame_1);
  dilation<<<grid_size, block_size>>>(mos_d_frame_1, mos_d_frame_2);

  err = hipMemcpy(red_frame.ptr(), mos_d_frame_2, frame_size,
                   hipMemcpyDeviceToHost);
  (err != hipSuccess) ? printf("CUDA err: %s\n", hipGetErrorString(err)) : 0;

  std::vector<Blob> blobs;
  int num_blobs = -2;
  num_blobs = get_blobs(red_frame, blobs);
  std::vector<Pt> blob_centres;
  for (size_t i = 0; i < blobs.size(); i++) {
    blob_centres.push_back({blobs[i].cen_x, blobs[i].cen_y});
  }

  if (blob_centres.size() == 0) {
    blob_centres.push_back({-1, -1});
  }

  for (size_t i = 0; i < blobs.size(); i++) {
    cv::circle(red_frame, cv::Point(blobs[i].cen_x, blobs[i].cen_y), 20,
               cv::Scalar(150, 255, 255), 2);
    cv::putText(red_frame, std::to_string(i),
                cv::Point(blobs[i].cen_x + 10, blobs[i].cen_y + 10),
                cv::FONT_HERSHEY_SIMPLEX, 1, cv::Scalar(255, 255, 255), 1);
  }

  cv::putText(red_frame, "num blobs = " + std::to_string(num_blobs),
              cv::Point(10, 30), cv::FONT_HERSHEY_SIMPLEX, 1,
              cv::Scalar(255, 255, 255), 2);
  cv::imshow("mosquitoes", red_frame);
  cv::waitKey(1);
  return blob_centres;
}

}  // namespace gpu
